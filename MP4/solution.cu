#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "CUDA error: ", hipGetErrorString(err));              \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

const int MASK_WIDTH = 3;
const int MASK_RADIUS = MASK_WIDTH / 2;
const int TILE_SIZE = 4;
const int BLOCK_SIZE = TILE_SIZE + MASK_WIDTH - 1;

//@@ Define constant memory for device kernel here
__constant__ float deviceKernel[MASK_WIDTH][MASK_WIDTH][MASK_WIDTH];

__global__ void conv3d(float *A, float *B, const int z_size,
                       const int y_size, const int x_size) {
  //@@ Insert kernel code here

  __shared__ float A_s[BLOCK_SIZE][BLOCK_SIZE][BLOCK_SIZE];

  const int out_z = blockIdx.z * TILE_SIZE + threadIdx.z;
  const int out_y = blockIdx.y * TILE_SIZE + threadIdx.y;
  const int out_x = blockIdx.x * TILE_SIZE + threadIdx.x;

  const int in_z = out_z - MASK_RADIUS;
  const int in_y = out_y - MASK_RADIUS;
  const int in_x = out_x - MASK_RADIUS;

  float output = 0.0;

  if (in_z >= 0 && in_z < z_size && in_y >= 0 && in_y < y_size &&
      in_x >= 0 && in_x < x_size) {
    A_s[threadIdx.z][threadIdx.y][threadIdx.x] =
        A[in_z * (y_size * x_size) + in_y * x_size + in_x];
  } else {
    A_s[threadIdx.z][threadIdx.y][threadIdx.x] = 0.0f;
  }
  __syncthreads();

  if (threadIdx.z < TILE_SIZE && threadIdx.y < TILE_SIZE &&
      threadIdx.x < TILE_SIZE) {
    for (int z = 0; z < MASK_WIDTH; ++z) {
      for (int y = 0; y < MASK_WIDTH; ++y) {
        for (int x = 0; x < MASK_WIDTH; ++x) {
          output += deviceKernel[z][y][x] *
                    A_s[z + threadIdx.z][y + threadIdx.y][x + threadIdx.x];
        }
      }
    }

    if (out_z < z_size && out_y < y_size && out_x < x_size) {
      B[out_z * (y_size * x_size) + out_y * x_size + out_x] = output;
    }
  }
}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int z_size;
  int y_size;
  int x_size;
  int inputLength, kernelLength;
  float *hostInput;
  float *hostKernel;
  float *hostOutput;
  float *deviceInput;
  float *deviceOutput;

  args = wbArg_read(argc, argv);

  // Import data
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostKernel =
      (float *)wbImport(wbArg_getInputFile(args, 1), &kernelLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));

  // First three elements are the input dimensions
  z_size = hostInput[0];
  y_size = hostInput[1];
  x_size = hostInput[2];
  wbLog(TRACE, "The input size is ", z_size, "x", y_size, "x", x_size);
  assert(z_size * y_size * x_size == inputLength - 3);
  assert(kernelLength == 27);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  //@@ Allocate GPU memory here
  hipMalloc((void **)&deviceInput, (inputLength - 3) * sizeof(float));
  hipMalloc((void **)&deviceOutput, (inputLength - 3) * sizeof(float));
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  //@@ Copy input and kernel to GPU here
  // Recall that the first three elements of hostInput are dimensions and
  // do
  // Not need to be copied to the gpu
  wbCheck(hipMemcpy(deviceInput,
                     &hostInput[3], // first three elements are dimensions
                     (inputLength - 3) * sizeof(float),
                     hipMemcpyHostToDevice););
  wbCheck(hipMemcpyToSymbol(HIP_SYMBOL(deviceKernel), hostKernel,
                             MASK_WIDTH * MASK_WIDTH * MASK_WIDTH *
                                 sizeof(float)););
  wbTime_stop(Copy, "Copying data to the GPU");

  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ Initialize grid and block dimensions here
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(ceil(((float)x_size) / TILE_SIZE),
               ceil(((float)y_size) / TILE_SIZE),
               ceil(((float)z_size) / TILE_SIZE));

  //@@ Launch the GPU kernel here
  conv3d<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, z_size, y_size,
                                x_size);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Doing the computation on the GPU");

  wbTime_start(Copy, "Copying data from the GPU");
  //@@ Copy the device memory back to the host here
  // Recall that the first three elements of the output are the dimensions
  // and should not be set here (they are set below)
  hipMemcpy(&hostOutput[3], deviceOutput,
             (inputLength - 3) * sizeof(float), hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  // Set the output dimensions for correctness checking
  hostOutput[0] = z_size;
  hostOutput[1] = y_size;
  hostOutput[2] = x_size;
  wbSolution(args, hostOutput, inputLength);

  // Free device memory
  hipFree(deviceInput);
  hipFree(deviceOutput);

  // Free host memory
  free(hostInput);
  free(hostOutput);
  return 0;
}
